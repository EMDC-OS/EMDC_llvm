/* Copyright (c) 2019-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <iostream>
#include <atomic>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>
#include <string>
#include <boost/interprocess/managed_shared_memory.hpp>
#include <cstdlib>

#define MEM_SIZE 1024

using namespace std::string_literals;
namespace bip = boost::interprocess;

static_assert(ATOMIC_INT_LOCK_FREE == 2, "atomic_int must be lock-free");


static inline void
checkRtError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << hipGetErrorString(res) << std::endl;
        abort();
    }
}

#define CHECK_RT(x) checkRtError(x, #x, __FILE__, __LINE__);

static inline void
checkDrvError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        const char *errStr = NULL;
        (void)hipDrvGetErrorString(res, &errStr);
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << errStr << std::endl;
        abort();
    }
}

#define CHECK_DRV(x) checkDrvError(x, #x, __FILE__, __LINE__);

class MMAPAllocation {
    size_t sz;
    hipMemGenericAllocationHandle_t hdl;
    hipMemAccessDesc accessDesc;
    hipDeviceptr_t ptr;
public:
    MMAPAllocation(size_t size, int dev = 0) {
        size_t aligned_sz;
        hipMemAllocationProp prop = {};
        prop.type = hipMemAllocationTypePinned;
        prop.location.type = hipMemLocationTypeDevice;
        prop.location.id = dev;
        accessDesc.location = prop.location;
        accessDesc.flags = hipMemAccessFlagsProtReadWrite;

        //CHECK_DRV(hipMemGetAllocationGranularity(&aligned_sz, &prop, hipMemAllocationGranularityRecommended));
        CHECK_DRV(hipMemGetAllocationGranularity(&aligned_sz, &prop, hipMemAllocationGranularityMinimum));
        sz = ((size + aligned_sz - 1) / aligned_sz) * aligned_sz;

        CHECK_DRV(hipMemCreate(&hdl, sz, &prop, 0));
        CHECK_DRV(hipMemAddressReserve(&ptr, sz, 0ULL, 0ULL, 0ULL));
        CHECK_DRV(hipMemMap(ptr, sz, 0ULL, hdl, 0ULL));

		bip::managed_shared_memory segment3(bip::open_only, "szMem2");
		auto bar1 = segment3.find<std::atomic_int>("the barrier2");
		--*bar1.first;
		std::cout << "The barreir count: "<< *bar1.first<<"\n";
		while(1){
			if(*bar1.first ==0)	break;	
		}
	
        CHECK_DRV(hipMemSetAccess(ptr, sz, &accessDesc, 1ULL));
    }
    ~MMAPAllocation() {
        CHECK_DRV(hipMemUnmap(ptr, sz));
        CHECK_DRV(hipMemAddressFree(ptr, sz));
        CHECK_DRV(hipMemRelease(hdl));
    }
};



int main(int argc, char** argv)
{
    //const size_t N = 4ULL;
    int supportsVMM = 0;
    hipDevice_t dev;

	if(argc == 2){ // parent process
		int num_proc = atoi(argv[1]);

		struct shm_remove{
			shm_remove(){ 
				bip::shared_memory_object::remove("szMem"); 
				bip::shared_memory_object::remove("szMem2");
				bip::shared_memory_object::remove("szMem3");
			}	
			~shm_remove(){
				 bip::shared_memory_object::remove("szMem"); 
				 bip::shared_memory_object::remove("szMem2");
				 bip::shared_memory_object::remove("szMem3");
			}	
		}remover;

		bip::managed_shared_memory segment(bip::create_only, "szMem", MEM_SIZE);
		auto ap = segment.construct<std::atomic_int>("the barrier")(num_proc);

		bip::managed_shared_memory segment2(bip::create_only, "szMem2", MEM_SIZE);
		auto ap2 = segment2.construct<std::atomic_int>("the barrier2")(num_proc);

		bip::managed_shared_memory segment3(bip::create_only, "szMem3", MEM_SIZE);
		auto ap3 = segment3.construct<std::atomic_int>("the barrier3")(num_proc);

		std::cout << "The leader counter : "<<*ap<<"\n";
		while(1){
			if(*ap == 0){
				break;
			}	
		}
		std::cout << "hipMemCreate Destroy barrier\n";

		std::cout << "The leader counter : "<<*ap2<<"\n";
		while(1){
			if(*ap2 == 0){
				break;
			}	
		}
		std::cout << "hipMemSetAccess Destroy barrier\n";

		std::cout << "The leader counter : "<<*ap2<<"\n";
		while(1){
			if(*ap3 == 0){
				break;
			}	
		}
		std::cout << "hipMemRelease Destroy barrier\n";

		segment.destroy<std::atomic_int>("the barrier");
		segment2.destroy<std::atomic_int>("the barrier2");
		segment3.destroy<std::atomic_int>("the barrier3");

	}
	else if(argc == 3){

		size_t ALLOC_SIZE = 4096ULL << atoi(argv[2]);

		std::cout<<"Allocation size : "<<ALLOC_SIZE<<"\n";

		CHECK_RT(hipFree(0));  // Force and check the initialization of the runtime

		CHECK_DRV(hipCtxGetDevice(&dev));
		CHECK_DRV(hipDeviceGetAttribute(&supportsVMM, CU_DEVICE_ATTRIBUTE_VIRTUAL_ADDRESS_MANAGEMENT_SUPPORTED, dev));
		int *x = nullptr;

		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

		CHECK_RT(hipMalloc(&x, 4096));
		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

		std::cout<<"hipMalloc elapsed \t\t"<<std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count()<<"us \n";

		CHECK_RT(hipFree(x));

		/* decrease barrier by 1 */
		bip::managed_shared_memory segment1(bip::open_only, "szMem");
		auto bar = segment1.find<std::atomic_int>("the barrier");
		--*bar.first;
		std::cout << "The barreir count: "<< *bar.first<<"\n";
		while(1){
			if(*bar.first ==0)	break;	
		}
		/* all threads are synchronized  */

		if (supportsVMM) {
			// Now use the Virtual Memory Management APIs
			begin = std::chrono::steady_clock::now();
			MMAPAllocation *allocMMAP = new MMAPAllocation(ALLOC_SIZE);
			end = std::chrono::steady_clock::now();
			std::cout<<"MMAP Allocation elapsed \t"<<std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count()<<"us \n";

			bip::managed_shared_memory segment3(bip::open_only, "szMem3");
			auto bar3 = segment3.find<std::atomic_int>("the barrier3");
			--*bar3.first;
			std::cout << "The barreir count: "<< *bar3.first<<"\n";
			while(1){
				if(*bar3.first ==0)	break;	
			}
			delete(allocMMAP);
		}

		/*
		   hipDeviceptr_t ptr;
		   begin = std::chrono::steady_clock::now();
		CHECK_DRV(hipMalloc(&ptr, ALLOC_SIZE));
		end = std::chrono::steady_clock::now();
		std::cout<<"hipMalloc elapsed \t"<<std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count()<<"us \n";
		CHECK_DRV(hipFree(ptr));
		*/
	}
	else{
		;	
	}
	return 0;
}
