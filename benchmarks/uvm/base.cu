#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cstring>

namespace chr = std::chrono;

inline void init_arr(float* arr){
	unsigned long long limit = (sizeof arr )/4;
	for(unsigned long long i=0; i<limit; ++i){
		arr[i] = 0.0f;	
	}
}

int main(){
	float *h_arr;
	float *d_arr;
	float *uvm_arr;

	hipFree(0);

	unsigned long long alloc_size=2*1024*1024; // 2MB	
	for( int i=0; i<=13; ++i, alloc_size <<= 1){
		std::cout<<"alloc size: "<<alloc_size/1024/1024<<"(MB)\n";
		chr::steady_clock::time_point begin = chr::steady_clock::now();
		h_arr = (float*)malloc(alloc_size);
		init_arr(h_arr);
		hipMalloc((void**)&d_arr, alloc_size);
		hipMemcpy(d_arr,h_arr,alloc_size,hipMemcpyDefault);
		hipMemcpy(h_arr,d_arr,alloc_size,hipMemcpyDefault);
		init_arr(h_arr);
		hipFree(d_arr);
		free(h_arr);
		chr::steady_clock::time_point end = chr::steady_clock::now();
		std::cout<<"malloc:\t\t\t"<<chr::duration_cast<chr::microseconds>(end-begin).count()<<"\tus\n";

		begin = chr::steady_clock::now();
		hipHostMalloc((void**)&h_arr,alloc_size);
		init_arr(h_arr);
		hipMalloc((void**)&d_arr, alloc_size);
		hipMemcpy(d_arr,h_arr,alloc_size,hipMemcpyHostToDevice);
		hipMemcpy(h_arr,d_arr,alloc_size,hipMemcpyDeviceToHost);
		init_arr(h_arr);
		hipFree(d_arr);
		hipHostFree(h_arr);
		end = chr::steady_clock::now();
		std::cout<<"hipHostMalloc:\t\t"<<chr::duration_cast<chr::microseconds>(end-begin).count()<<"\tus\n";

		begin = chr::steady_clock::now();
		hipMallocManaged((void**)&uvm_arr, alloc_size);
		init_arr(uvm_arr);
		hipMemAdvise(uvm_arr, alloc_size, hipMemAdviseSetPreferredLocation,0);
		hipMemAdvise(uvm_arr, alloc_size, hipMemAdviseSetAccessedBy, hipCpuDeviceId);
		hipMemPrefetchAsync(uvm_arr, alloc_size, 0);
//		hipMemPrefetchAsync(uvm_arr, alloc_size, hipCpuDeviceId);
		init_arr(uvm_arr);
		hipFree(uvm_arr);
		end = chr::steady_clock::now();
		std::cout<<"uvm:\t\t\t"<<chr::duration_cast<chr::microseconds>(end-begin).count()<<"\tus\n";
	}


	
	return 0;	
}
