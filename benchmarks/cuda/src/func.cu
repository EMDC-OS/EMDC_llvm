#include "hip/hip_runtime.h"

#include <stdio.h>


__global__ void addKernel(float *c, const float *a, const float *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

hipError_t func1( hipFuncAttributes* attrib, void (*ptr)() )
{
    return hipFuncGetAttributes(attrib, reinterpret_cast<const void*>(ptr));
}

hipError_t func2( hipFuncAttributes* attrib, const char* ptr )
{
    return hipFuncGetAttributes(attrib, reinterpret_cast<const void*>(ptr));
}


hipError_t func2( hipFuncAttributes* attrib, float* ptr )
{
    return func2( attrib, (const char*) ptr);
}

int main()
{
    hipFuncAttributes attrib;
    hipError_t err;

    void (*ptr2)() = (void(*)())(addKernel);  // OK on Visual Studio
  //  err = func1(&attrib, ptr2);
  //  printf("result: %s, reg1: %d\n", cudaGetErrorString(err), attrib.numRegs);
    hipFuncGetAttributes(&attrib, reinterpret_cast<const void*>(ptr2));
    printf("result: %s, reg1: %d\n", hipGetErrorString(err), attrib.numRegs);
 
}