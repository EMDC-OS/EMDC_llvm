#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string>
#include <iostream>

using namespace std;

static inline void
checkRtError(hipError_t res, const char *tok, const char *file, unsigned line)
{
	if (res != hipSuccess) {
		std::cerr << file << ':' << line << ' ' << tok
			<< "failed (" << (unsigned)res << "): " << hipGetErrorString(res) << std::endl;
		abort();
	}
}

#define CHECK_RT(x) checkRtError(x, #x, __FILE__, __LINE__);

#ifdef DEBUG
inline void print_k(string msg){
	string cmd = "echo 'HSI:"+ msg +"'| sudo tee /dev/kmsg";
	system(cmd.c_str());
}
#else
	#define print_k(x) ;
#endif

__global__ void foo(float* in, float* in2, float* output){


}

int main(){
	int device_id, attr_val, num_devices;
	hipDeviceProp_t prop;
	float* arr;
	hipFuncCache_t cacheConfig;
	hipMemPool_t memPool;
	size_t limitValue;
	int P2PAttr;
	hipSharedMemConfig SHMEMConfig;
	int leastPri, greatestPri;
	unsigned int device_flags;
	//size_t maxWidthInElements;
	//hipChannelFormatDesc cfmt={1,1,1,1};
	//void nvSciSyncAttrList;
	
	/* init CUDA runtime */
	print_k("Init CUDA");
	hipFree(0);
	print_k("Init CUDA done!");
	
	print_k("Analysis begin");
	
	/* start of Device Management */
	print_k("hipGetDeviceProperties");
	CHECK_RT(	hipGetDeviceProperties(&prop, 0)		);
	print_k("hipChooseDevice");
	CHECK_RT(	hipChooseDevice(&device_id, &prop)		);	
	print_k("hipDeviceGetAttribute");
	CHECK_RT(	hipDeviceGetAttribute(&attr_val, hipDeviceAttributeMemoryPoolsSupported,0)	);
	print_k("hipDeviceGetCacheConfig");
	CHECK_RT(	hipDeviceGetCacheConfig(&cacheConfig)	);
	print_k("hipDeviceGetDefaultMemPool");
	CHECK_RT(	hipDeviceGetDefaultMemPool(&memPool, 0)	);
	print_k("hipDeviceGetLimit");
	CHECK_RT(	hipDeviceGetLimit(&limitValue, hipLimitStackSize)	);
	//print_k("cudaDeviceGetNvSciSyncAttributes");
	//CHECK_RT(	cudaDeviceGetNvSciSyncAttributes(&nvSciSyncAttrList, 0, cudaNvSciSyncAttrSignal) 	);
	print_k("hipDeviceGetP2PAttribute");
	CHECK_RT(	hipDeviceGetP2PAttribute(&P2PAttr, hipDevP2PAttrAccessSupported, 0, 1)		);
	//hipDeviceGetPCIBusId 
	print_k("hipDeviceGetSharedMemConfig");
	CHECK_RT(	hipDeviceGetSharedMemConfig(&SHMEMConfig)	);
	print_k("hipDeviceGetStreamPriorityRange");
	CHECK_RT(	hipDeviceGetStreamPriorityRange (&leastPri, &greatestPri)		);
	//print_k("hipDeviceGetTexture1DLinearMaxWidth");
	//CHECK_RT(	hipDeviceGetTexture1DLinearMaxWidth(&maxWidthInElements, &cfmt,0)	);
	print_k("hipDeviceReset");
	CHECK_RT(	hipDeviceReset()	);
	print_k("hipDeviceSetCacheConfig");
	CHECK_RT(	hipDeviceSetCacheConfig(hipFuncCachePreferShared)	);	//hipFuncCachePreferNone: no preference for shared memory or L1 (default); hipFuncCachePreferShared: prefer larger shared memory and smaller L1 cache;hipFuncCachePreferL1: prefer larger L1 cache and smaller shared memory;hipFuncCachePreferEqual: prefer equal size L1 cache and shared memory
	print_k("hipDeviceSetLimit");
	CHECK_RT(	hipDeviceSetLimit(hipLimitStackSize,512) 		);
	print_k("hipDeviceSetSharedMemConfig");
	CHECK_RT(	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte) 	);
	print_k("hipGetDevice");
	CHECK_RT(	hipGetDevice(&device_id)	);
	print_k("hipGetDeviceCount");
	CHECK_RT(	hipGetDeviceCount(&num_devices)	);
	print_k("hipGetDeviceFlags");
	CHECK_RT(	hipGetDeviceFlags(&device_flags)	);

	print_k("hipSetDevice");
	CHECK_RT(	hipSetDevice(0) 	);
	print_k("hipSetDeviceFlags");
	CHECK_RT(	hipSetDeviceFlags(hipDeviceScheduleAuto) 		);
	/* end of Device Management*/

	/* start of Error Handling */
	hipError_t lastError;
	print_k("hipGetLastError");	
	CHECK_RT(	lastError = hipGetLastError()		);
	print_k("hipPeekAtLastError");
	CHECK_RT(	lastError = hipPeekAtLastError()	);
	/* end of Error Handling */

	/* start of Stream Management */
	hipStream_t hStream, hStreamFlags, hStreamPri;
	hipLaunchAttributeValue streamAttrVal;
	unsigned int streamFlags;
	int streamPri;

	print_k("cudaCtxResetPersistingL2Cache");
	CHECK_RT(	cudaCtxResetPersistingL2Cache()		);
	print_k("hipStreamCreate");
	CHECK_RT(	hipStreamCreate(&hStream)		);
	print_k("hipStreamCreateWithFlags");
	CHECK_RT(	hipStreamCreateWithFlags(&hStreamFlags, hipStreamNonBlocking)		);
	print_k("hipStreamCreateWithPriority");
	CHECK_RT(	hipStreamCreateWithPriority(&hStreamPri, hipStreamDefault, 1)		);
	print_k("cudaStreamCopyAttributes"); // src, dst
	CHECK_RT(	cudaStreamCopyAttributes(hStream, hStreamFlags)		);
	print_k("hipStreamDestroy");
	CHECK_RT(	hipStreamDestroy(hStream)		);
	//print_k("hipStreamGetAttribute");
	//CHECK_RT(	hipStreamGetAttribute(hStreamPri, hipAccessPolicyWindow.num_bytes, &streamAttrVal)		);
	print_k("hipStreamGetFlags");
	CHECK_RT(	hipStreamGetFlags(hStreamPri, &streamFlags)		);
	print_k("hipStreamGetPriority");
	CHECK_RT(	hipStreamGetPriority(hStreamFlags, &streamPri)		);
	print_k("hipStreamQuery");
	CHECK_RT(	hipStreamQuery(hStreamPri)		);
	print_k("hipStreamSynchronize");
	CHECK_RT(	hipStreamSynchronize(hStreamPri)		);
	/* end of Stream Management */

	/* start of Event Management */
	hipEvent_t event, eventFlags;

	print_k("hipEventCreate");
	CHECK_RT(	hipEventCreate(&event)		);
	print_k("hipEventCreateWithFlags");
	CHECK_RT(	hipEventCreateWithFlags(&eventFlags, hipEventDefault)		);
	print_k("hipEventDestroy");
	CHECK_RT(	hipEventDestroy(eventFlags)		);
	print_k("hipEventRecord");
	CHECK_RT(	hipEventRecord(event)		);
	print_k("hipEventQuery");
	CHECK_RT(	hipEventQuery(event)		);
	print_k("hipEventSynchronize");
	CHECK_RT(	hipEventSynchronize(event)		);
	//print_k("hipEventElapsedTime");
	//CHECK_RT(	hipEventElapsedTime()	);
	/* end of Event Management */

	/* start of External REsource Interoperability */
	/* end of External REsource Interoperability */

	/* start of Execution Control */
	hipFuncAttributes funcAttr;
	void (*p_foo)() = (void(*)())(foo);
	print_k("hipFuncGetAttributes");
	CHECK_RT(	hipFuncGetAttributes(&funcAttr, reinterpret_cast<const void*>(p_foo ))		);
	print_k("hipFuncSetAttribute\tSharedMemoryCarveout");
	CHECK_RT(	hipFuncSetAttribute(reinterpret_cast<const void*>(p_foo), hipFuncAttributePreferredSharedMemoryCarveout, 1)		);
	print_k("hipFuncSetAttribute\tMaxDynamicSharedMemorySize");
	CHECK_RT(	hipFuncSetAttribute(reinterpret_cast<const void*>(p_foo), hipFuncAttributeMaxDynamicSharedMemorySize, 1024*64)		);	
	print_k("hipFuncSetCacheConfig\tPreferL1");
	CHECK_RT(	hipFuncSetCacheConfig(reinterpret_cast<const void*>(p_foo), hipFuncCachePreferL1)		);
	print_k("hipFuncSetSharedMemConfig");
	CHECK_RT(	hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(p_foo), hipSharedMemBankSizeDefault)		);
	//print_k("hipLaunchCooperativeKernel");	
	//CHECK_RT(	hipLaunchCooperativeKernel(reinterpret_cast<const void*>(p_foo), (1,1,1), (1,1,1), (a,b,c), )		);
	/* end of Execution Control */

	/* start of Occupancy */
	size_t dynamicSmemSize;
	int numBlocks, numClusters;
	hipLaunchConfig_t launchConfig;

	print_k("cudaOccupancyAvailableDynamicSMemPerBlock");
	CHECK_RT(	cudaOccupancyAvailableDynamicSMemPerBlock(&dynamicSmemSize,foo,1,1)		);
	print_k("hipOccupancyMaxActiveBlocksPerMultiprocessor");
	CHECK_RT(	hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, p_foo, 1, dynamicSmemSize)		);
	print_k("hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags");
	CHECK_RT(	hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(&numBlocks, p_foo, 256, dynamicSmemSize, hipOccupancyDisableCachingOverride)		);
	//print_k("cudaOccupancyMaxActiveClusters");
	//CHECK_RT(	cudaOccupancyMaxActiveClusters(&numClusters, p_foo, &launchConfig)		);			not supported
	/* end of Occupancy */

	/* start of Memory Management */
	float *hArr, *uvmArr;
	unsigned int hArrFlags;

	print_k("hipHostAlloc");
	CHECK_RT(	hipHostAlloc((void**)&hArr, 1024*1024*4, hipHostMallocDefault)		);
	print_k("hipHostGetFlags");
	CHECK_RT(	hipHostGetFlags(&hArrFlags, hArr)		);
	print_k("hipHostFree");
	CHECK_RT(	hipHostFree(hArr)		);
	hArr = (float*)malloc(1024*1024*4);
	print_k("hipHostRegister");
	CHECK_RT(	hipHostRegister(hArr, 1024*1024*4, hipHostRegisterDefault)		);
	print_k("hipHostUnregister");
	CHECK_RT(	hipHostUnregister(hArr)		);
	hipPitchedPtr dPitchedArr;
	hipExtent	extend3D = make_hipExtent(1024,1024,4); 
	print_k("hipMalloc3D");
	CHECK_RT(	hipMalloc3D(&dPitchedArr, extend3D)		);
	//hipArray_t dArray;
	//hipChannelFormatDesc arrDesc; arrDesc.x=1024; arrDesc.y=1; arrDesc.z=1; arrDesc.w=1; arrDesc.f=hipChannelFormatKindFloat;
	//print_k("hipMallocArray");
	//CHECK_RT(	hipMallocArray(&dArray, &arrDesc, 1024, 0, 0)		);
	print_k("hipMallocManaged");
	CHECK_RT(	hipMallocManaged((void**)&uvmArr, 1024*1024*256)		);
	print_k("hipMemAdvise SetPreferredLocation");
	CHECK_RT(	hipMemAdvise(uvmArr, 1024*1024*256, hipMemAdviseSetPreferredLocation, 0)		);
	print_k("hipMemAdvise SetAccessedBy");
	CHECK_RT(	hipMemAdvise(uvmArr, 1024*1024*256, hipMemAdviseSetAccessedBy, 1)		);
	print_k("hipMemGetInfo");
	size_t memFree, memTotal;
	CHECK_RT(	hipMemGetInfo(&memFree, &memTotal)		);
	print_k("hipMalloc");
	CHECK_RT(	hipMalloc((void**)&arr, sizeof(float)*4096)		);
	print_k("hipMemcpy");
	CHECK_RT(	hipMemcpy(arr,hArr, 1024, hipMemcpyDefault)		);
	print_k("hipMemcpyAsync Pri");
	CHECK_RT(	hipMemcpyAsync(arr,hArr, 1024, hipMemcpyDefault, hStreamPri)		);
	print_k("hipMemcpyAsync 0");
	CHECK_RT(	hipMemcpyAsync(arr,hArr, 1024, hipMemcpyDefault, 0)		);
	print_k("hipMemset");
	CHECK_RT(	hipMemset(arr, 0, 1024)	);
	print_k("hipMemsetAsync Pri");
	CHECK_RT(	hipMemsetAsync(arr, 1, 1024, hStreamPri)	);
	print_k("hipMemsetAsync 0");
	CHECK_RT(	hipMemsetAsync(arr, 1, 1024, 0)	);
	print_k("hipFree");
	CHECK_RT(	hipFree(arr)		);
	/* end of Memory Management */
	
	/* start of Stream Ordered Memory Allocator */
	print_k("hipMemPoolCreate");
	hipMemPool_t mem_pool; hipMemPoolProps poolProps; memset(&poolProps, 0, sizeof(hipMemPoolProps)); 
	poolProps.allocType=hipMemAllocationTypePinned; poolProps.handleTypes=hipMemHandleTypePosixFileDescriptor; poolProps.location.type=hipMemLocationTypeDevice; poolProps.location.id=0;

	CHECK_RT(	hipMemPoolCreate(&mem_pool, &poolProps)		);
	float *dArrPool;
	print_k("hipMallocFromPoolAsync");
	CHECK_RT(	hipMallocFromPoolAsync((void**)&dArrPool,1024*4, memPool, hStreamPri)		);
	print_k("hipFreeAsync");
	CHECK_RT(	hipFreeAsync(dArrPool,	hStreamPri)		);
	print_k("hipMallocFromPoolAsync");
	CHECK_RT(	hipMallocFromPoolAsync((void**)&dArrPool,1024*4, memPool, hStreamPri)		);	
	print_k("hipMallocAsync");
	CHECK_RT(	hipMallocAsync((void**)&arr, 1024*1024*4, hStreamPri)		);
	/* end of Stream Ordered Memory Allocator */

	/* start of Texture Object Management */
	/* end of Texture Object Management */

	/* start of Surface Object Management */
	/* end of Surface Object Management */



	
	print_k("hipDeviceSynchronize");
	CHECK_RT(	hipDeviceSynchronize()		);
	print_k("Analysis end");
	return 0;	
}
