#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <unistd.h>
#include <nvml.h>


#define GB_2 2147483648
#define GB_4 4294967296

int main(){
	hipFreeArray(0);

	float *arr, *arr_h;
//	cudaMallocHost((void**)&arr_h, GB_2);
	arr_h = (float*)malloc(GB_2);
	//cudaError_t error = cudaMalloc((void**)&arr, 1024*1024*1024);
/*
	nvmlDevice_t dev_id_nvml;
	nvmlReturn_t ret_nvml;
	nvmlInit();

	ret_nvml = nvmlDeviceGetHandleByIndex(0,&dev_id_nvml);
	nvmlMemory_t mem_info;
	ret_nvml = nvmlDeviceGetMemoryInfo(dev_id_nvml, &mem_info);
	if(ret_nvml == NVML_SUCCESS)
		std::cout<<"before allocation Used gmem: "<<mem_info.used/1024/1024<<"\n";
*/
	hipError_t error = hipMalloc((void**)&arr, GB_2);
	if(error != hipSuccess){
		std::cout<< hipGetErrorString(error);
	}
/*
	ret_nvml = nvmlDeviceGetMemoryInfo(dev_id_nvml, &mem_info);
	if(ret_nvml == NVML_SUCCESS)
		std::cout<<"after allocation Used gmem: "<<mem_info.used/1024/1024<<"\n";
*/

	hipMemcpy(arr, arr_h, 1024*1024*1024, hipMemcpyHostToDevice);
//	cudaMemcpy(arr, arr_h, 512*1024*1024, cudaMemcpyHostToDevice);

//	nvmlShutdown();

	free(arr_h);
//	cudaFree(arr_h);
	hipFree(arr);
	
	return 0;	
}
